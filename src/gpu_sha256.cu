#include "hip/hip_runtime.h"
#include <stddef.h>
#include <stdint.h>
#include <string.h>

#include "sha256.h"
#include "utils.h"
#include <iostream>
using namespace std;

#ifdef SHA256

const __constant__ u32 __align__(8) cons_K256[64]
    = {0x428a2f98UL, 0x71374491UL, 0xb5c0fbcfUL, 0xe9b5dba5UL, 0x3956c25bUL, 0x59f111f1UL,
       0x923f82a4UL, 0xab1c5ed5UL, 0xd807aa98UL, 0x12835b01UL, 0x243185beUL, 0x550c7dc3UL,
       0x72be5d74UL, 0x80deb1feUL, 0x9bdc06a7UL, 0xc19bf174UL, 0xe49b69c1UL, 0xefbe4786UL,
       0x0fc19dc6UL, 0x240ca1ccUL, 0x2de92c6fUL, 0x4a7484aaUL, 0x5cb0a9dcUL, 0x76f988daUL,
       0x983e5152UL, 0xa831c66dUL, 0xb00327c8UL, 0xbf597fc7UL, 0xc6e00bf3UL, 0xd5a79147UL,
       0x06ca6351UL, 0x14292967UL, 0x27b70a85UL, 0x2e1b2138UL, 0x4d2c6dfcUL, 0x53380d13UL,
       0x650a7354UL, 0x766a0abbUL, 0x81c2c92eUL, 0x92722c85UL, 0xa2bfe8a1UL, 0xa81a664bUL,
       0xc24b8b70UL, 0xc76c51a3UL, 0xd192e819UL, 0xd6990624UL, 0xf40e3585UL, 0x106aa070UL,
       0x19a4c116UL, 0x1e376c08UL, 0x2748774cUL, 0x34b0bcb5UL, 0x391c0cb3UL, 0x4ed8aa4aUL,
       0x5b9cca4fUL, 0x682e6ff3UL, 0x748f82eeUL, 0x78a5636fUL, 0x84c87814UL, 0x8cc70208UL,
       0x90befffaUL, 0xa4506cebUL, 0xbef9a3f7UL, 0xc67178f2UL}; // __align__

__device__ uint32_t dev_load_bigendian_32(const uint8_t* x) {
    return (uint32_t) (x[3]) | (((uint32_t) (x[2])) << 8) | (((uint32_t) (x[1])) << 16)
        | (((uint32_t) (x[0])) << 24);
} // dev_load_bigendian_32

__device__ uint64_t dev_load_bigendian_64(const uint8_t* x) {
    return (uint64_t) (x[7]) | (((uint64_t) (x[6])) << 8) | (((uint64_t) (x[5])) << 16)
        | (((uint64_t) (x[4])) << 24) | (((uint64_t) (x[3])) << 32) | (((uint64_t) (x[2])) << 40)
        | (((uint64_t) (x[1])) << 48) | (((uint64_t) (x[0])) << 56);
} // dev_load_bigendian_64

__device__ void dev_store_bigendian_32(uint8_t* x, uint64_t u) {
    x[3] = (uint8_t) u;
    u >>= 8;
    x[2] = (uint8_t) u;
    u >>= 8;
    x[1] = (uint8_t) u;
    u >>= 8;
    x[0] = (uint8_t) u;
} // dev_store_bigendian_32

__device__ void dev_store_bigendian_64(uint8_t* x, uint64_t u) {
    x[7] = (uint8_t) u;
    u >>= 8;
    x[6] = (uint8_t) u;
    u >>= 8;
    x[5] = (uint8_t) u;
    u >>= 8;
    x[4] = (uint8_t) u;
    u >>= 8;
    x[3] = (uint8_t) u;
    u >>= 8;
    x[2] = (uint8_t) u;
    u >>= 8;
    x[1] = (uint8_t) u;
    u >>= 8;
    x[0] = (uint8_t) u;
} // dev_store_bigendian_64

#ifdef USING_SHA256_PTX

#if USING_SHA256_PTX_MODE == 0 // sign & default

#define Ch(a, b, c)                                                                                \
    ({                                                                                             \
        u32 result;                                                                                \
        asm("lop3.b32 %0, %1, %2, %3, 0xCA;" : "=r"(result) : "r"(a), "r"(b), "r"(c));             \
        result;                                                                                    \
    })

#define Maj(a, b, c)                                                                               \
    ({                                                                                             \
        u32 result;                                                                                \
        asm("lop3.b32 %0, %1, %2, %3, 0xE8;" : "=r"(result) : "r"(a), "r"(b), "r"(c));             \
        result;                                                                                    \
    })

#define ROL(v, n)                                                                                  \
    ({                                                                                             \
        u32 result;                                                                                \
        asm("shf.l.clamp.b32 %0, %1, %1, %2;\n\t" : "=r"(result) : "r"(v), "r"(n));                \
        result;                                                                                    \
    })

#define Sigma0_32(x)                                                                               \
    ({                                                                                             \
        u32 t1 = 0, t2 = 0;                                                                        \
        asm("shf.l.clamp.b32 %0, %2, %2, 30;\n\t"                                                  \
            "shf.l.clamp.b32 %1, %2, %2, 19;\n\t"                                                  \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            "shf.l.clamp.b32 %1, %2, %2, 10;\n\t"                                                  \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            : "+r"(t1), "+r"(t2)                                                                   \
            : "r"(x));                                                                             \
        t1;                                                                                        \
    })

#define Sigma1_32(x)                                                                               \
    ({                                                                                             \
        u32 t1 = 0, t2 = 0;                                                                        \
        asm("shf.l.clamp.b32 %0, %2, %2, 26;\n\t"                                                  \
            "shf.l.clamp.b32 %1, %2, %2, 21;\n\t"                                                  \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            "shf.l.clamp.b32 %1, %2, %2, 7;\n\t"                                                   \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            : "+r"(t1), "+r"(t2)                                                                   \
            : "r"(x));                                                                             \
        t1;                                                                                        \
    })

#define sigma0_32(x)                                                                               \
    ({                                                                                             \
        u32 t1 = 0, t2 = 0;                                                                        \
        asm("shf.l.clamp.b32 %0, %2, %2, 25;\n\t"                                                  \
            "shf.l.clamp.b32 %1, %2, %2, 14;\n\t"                                                  \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            "shr.b32 %1, %2, 3;\n\t"                                                               \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            : "+r"(t1), "+r"(t2)                                                                   \
            : "r"(x));                                                                             \
        t1;                                                                                        \
    })

#define sigma1_32(x)                                                                               \
    ({                                                                                             \
        u32 t1 = 0, t2 = 0;                                                                        \
        asm("shf.l.clamp.b32 %0, %2, %2, 15;\n\t"                                                  \
            "shf.l.clamp.b32 %1, %2, %2, 13;\n\t"                                                  \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            "shr.b32 %1, %2, 10;\n\t"                                                              \
            "xor.b32 %0, %0, %1;\n\t"                                                              \
            : "+r"(t1), "+r"(t2)                                                                   \
            : "r"(x));                                                                             \
        t1;                                                                                        \
    })

// __device__ __forceinline__ u32 Sigma1_32_Ch(u32 e, u32 f, u32 g) {
//     u32 t1 = 0, t2 = 0;
//     asm("shf.l.clamp.b32 %1, %0, %0, 26;\n\t"
//         "shf.l.clamp.b32 %2, %0, %0, 21;\n\t"
//         "xor.b32 %1, %1, %2;\n\t"
//         "shf.l.clamp.b32 %2, %0, %0, 7;\n\t"
//         "xor.b32 %1, %1, %2;\n\t"
//         "lop3.b32 %0, %0, %3, %4, 0xCA;\n\t"
//         "add.u32 %0, %0, %1;\n\t"
//         : "+r"(e), "+r"(t1), "+r"(t2)
//         : "r"(f), "r"(g));
//     return e;
// }

// __device__ __forceinline__ u32 Sigma0_32_Maj(u32 a, u32 b, u32 c) {
//     u32 t1 = 0, t2 = 0;
//     asm("shf.l.clamp.b32 %1, %0, %0, 26;\n\t"
//         "shf.l.clamp.b32 %2, %0, %0, 21;\n\t"
//         "xor.b32 %1, %1, %2;\n\t"
//         "shf.l.clamp.b32 %2, %0, %0, 7;\n\t"
//         "xor.b32 %1, %1, %2;\n\t"
//         "lop3.b32 %0, %0, %3, %4, 0xE8;\n\t"
//         "add.u32 %0, %0, %1;\n\t"
//         : "+r"(a), "+r"(t1), "+r"(t2)
//         : "r"(b), "r"(c));
//     return a;
// }

#elif USING_SHA256_PTX_MODE == 1 // kg

#define Ch(x, y, z) ((z) ^ ((x) & ((y) ^ (z))))
#define Maj(x, y, z) (((y) & ((x) | (z))) | ((x) & (z)))
#define ROL(v, n) (((v) << (n)) | ((v) >> (32 - (n))))

#define Sigma0_32(x) (ROL((x), 30) ^ ROL((x), 19) ^ ROL((x), 10))
#define Sigma1_32(x) (ROL((x), 26) ^ ROL((x), 21) ^ ROL((x), 7))
#define sigma0_32(x) (ROL((x), 25) ^ ROL((x), 14) ^ ((x) >> 3))
#define sigma1_32(x) (ROL((x), 15) ^ ROL((x), 13) ^ ((x) >> 10))

#elif USING_SHA256_PTX_MODE == 2 // verify
#define Ch(a, b, c)                                                                                \
    ({                                                                                             \
        u32 result;                                                                                \
        asm("lop3.b32 %0, %1, %2, %3, 0xCA;" : "=r"(result) : "r"(a), "r"(b), "r"(c));             \
        result;                                                                                    \
    })

#define Maj(a, b, c)                                                                               \
    ({                                                                                             \
        u32 result;                                                                                \
        asm("lop3.b32 %0, %1, %2, %3, 0xE8;" : "=r"(result) : "r"(a), "r"(b), "r"(c));             \
        result;                                                                                    \
    })

#define ROL(v, n) (((v) << (n)) | ((v) >> (32 - (n))))

#define Sigma0_32(x) (ROL((x), 30) ^ ROL((x), 19) ^ ROL((x), 10))
#define Sigma1_32(x) (ROL((x), 26) ^ ROL((x), 21) ^ ROL((x), 7))
#define sigma0_32(x) (ROL((x), 25) ^ ROL((x), 14) ^ ((x) >> 3))
#define sigma1_32(x) (ROL((x), 15) ^ ROL((x), 13) ^ ((x) >> 10))

#endif

#else // ifdef USING_SHA256_PTX

#define Ch(x, y, z) ((z) ^ ((x) & ((y) ^ (z))))
#define Maj(x, y, z) (((y) & ((x) | (z))) | ((x) & (z)))
#define ROL(v, n) (((v) << (n)) | ((v) >> (32 - (n))))
#define Sigma0_32(x) (ROL((x), 30) ^ ROL((x), 19) ^ ROL((x), 10))
#define Sigma1_32(x) (ROL((x), 26) ^ ROL((x), 21) ^ ROL((x), 7))
#define sigma0_32(x) (ROL((x), 25) ^ ROL((x), 14) ^ ((x) >> 3))
#define sigma1_32(x) (ROL((x), 15) ^ ROL((x), 13) ^ ((x) >> 10))

#endif // ifdef USING_SHA256_PTX

#ifdef USING_SHA256_INTEGER
#define HOST_c2l(c, l) (l = __byte_perm(*(c++), 0, 0x0123))
#else // ifdef USING_SHA256_INTEGER
#define HOST_c2l(c, l)                                                                             \
    (l = (((unsigned long) (*((c)++))) << 24), l |= (((unsigned long) (*((c)++))) << 16),          \
     l |= (((unsigned long) (*((c)++))) << 8), l |= (((unsigned long) (*((c)++)))))
#endif // ifdef USING_SHA256_INTEGER

#ifdef FASTER

#define ROUND_00_15(i, a, b, c, d, e, f, g, h)                                                     \
    T1 += h + Sigma1_32(e) + Ch(e, f, g) + cons_K256[i];                                           \
    h = Sigma0_32(a) + Maj(a, b, c);                                                               \
    d += T1;                                                                                       \
    h += T1;

#ifdef USING_SHA256_X_UNROLL
// x unroll version
__device__ void dev_crypto_hashblocks_sha256(uint8_t* __restrict__ statebytes,
                                             const void* __restrict__ in, size_t inlen) {

    u32 state[8];
    u32 a, b, c, d, e, f, g, h, s0, s1, T1;
    u32 X0, X1, X2, X3;
    u32 X4, X5, X6, X7;
    u32 X8, X9, X10, X11;
    u32 X12, X13, X14, X15;
    u32 num = inlen / 64;

    for (int i = 0; i < 8; i++)
        state[i] = dev_load_bigendian_32(statebytes + 4 * i);

#ifdef USING_SHA256_INTEGER
    const u32* data = (const u32*) in;
#else  // ifdef USING_SHA256_INTEGER
    const u8* data = (const u8*) in;
#endif // ifdef USING_SHA256_INTEGER

    while (num--) {
        a = state[0];
        b = state[1];
        c = state[2];
        d = state[3];
        e = state[4];
        f = state[5];
        g = state[6];
        h = state[7];

        u32 l;

        (void) HOST_c2l(data, l);
        T1 = X0 = l;
        ROUND_00_15(0, a, b, c, d, e, f, g, h);
        (void) HOST_c2l(data, l);
        T1 = X1 = l;
        ROUND_00_15(1, h, a, b, c, d, e, f, g);
        (void) HOST_c2l(data, l);
        T1 = X2 = l;
        ROUND_00_15(2, g, h, a, b, c, d, e, f);
        (void) HOST_c2l(data, l);
        T1 = X3 = l;
        ROUND_00_15(3, f, g, h, a, b, c, d, e);
        (void) HOST_c2l(data, l);
        T1 = X4 = l;
        ROUND_00_15(4, e, f, g, h, a, b, c, d);
        (void) HOST_c2l(data, l);
        T1 = X5 = l;
        ROUND_00_15(5, d, e, f, g, h, a, b, c);
        (void) HOST_c2l(data, l);
        T1 = X6 = l;
        ROUND_00_15(6, c, d, e, f, g, h, a, b);
        (void) HOST_c2l(data, l);
        T1 = X7 = l;
        ROUND_00_15(7, b, c, d, e, f, g, h, a);
        (void) HOST_c2l(data, l);
        T1 = X8 = l;
        ROUND_00_15(8, a, b, c, d, e, f, g, h);
        (void) HOST_c2l(data, l);
        T1 = X9 = l;
        ROUND_00_15(9, h, a, b, c, d, e, f, g);
        (void) HOST_c2l(data, l);
        T1 = X10 = l;
        ROUND_00_15(10, g, h, a, b, c, d, e, f);
        (void) HOST_c2l(data, l);
        T1 = X11 = l;
        ROUND_00_15(11, f, g, h, a, b, c, d, e);
        (void) HOST_c2l(data, l);
        T1 = X12 = l;
        ROUND_00_15(12, e, f, g, h, a, b, c, d);
        (void) HOST_c2l(data, l);
        T1 = X13 = l;
        ROUND_00_15(13, d, e, f, g, h, a, b, c);
        (void) HOST_c2l(data, l);
        T1 = X14 = l;
        ROUND_00_15(14, c, d, e, f, g, h, a, b);
        (void) HOST_c2l(data, l);
        T1 = X15 = l;
        ROUND_00_15(15, b, c, d, e, f, g, h, a);

        // #pragma unroll
        for (int i = 16; i < 64; i += 16) {
            s0 = sigma0_32(X1);
            s1 = sigma1_32(X14);
            T1 = X0 += s0 + s1 + X9;
            ROUND_00_15(i + 0, a, b, c, d, e, f, g, h);

            s0 = sigma0_32(X2);
            s1 = sigma1_32(X15);
            T1 = X1 += s0 + s1 + X10;
            ROUND_00_15(i + 1, h, a, b, c, d, e, f, g);

            s0 = sigma0_32(X3);
            s1 = sigma1_32(X0);
            T1 = X2 += s0 + s1 + X11;
            ROUND_00_15(i + 2, g, h, a, b, c, d, e, f);

            s0 = sigma0_32(X4);
            s1 = sigma1_32(X1);
            T1 = X3 += s0 + s1 + X12;
            ROUND_00_15(i + 3, f, g, h, a, b, c, d, e);

            s0 = sigma0_32(X5);
            s1 = sigma1_32(X2);
            T1 = X4 += s0 + s1 + X13;
            ROUND_00_15(i + 4, e, f, g, h, a, b, c, d);

            s0 = sigma0_32(X6);
            s1 = sigma1_32(X3);
            T1 = X5 += s0 + s1 + X14;
            ROUND_00_15(i + 5, d, e, f, g, h, a, b, c);

            s0 = sigma0_32(X7);
            s1 = sigma1_32(X4);
            T1 = X6 += s0 + s1 + X15;
            ROUND_00_15(i + 6, c, d, e, f, g, h, a, b);

            s0 = sigma0_32(X8);
            s1 = sigma1_32(X5);
            T1 = X7 += s0 + s1 + X0;
            ROUND_00_15(i + 7, b, c, d, e, f, g, h, a);

            // 8 - 16
            s0 = sigma0_32(X9);
            s1 = sigma1_32(X6);
            T1 = X8 += s0 + s1 + X1;
            ROUND_00_15(i + 8, a, b, c, d, e, f, g, h);

            s0 = sigma0_32(X10);
            s1 = sigma1_32(X7);
            T1 = X9 += s0 + s1 + X2;
            ROUND_00_15(i + 9, h, a, b, c, d, e, f, g);

            s0 = sigma0_32(X11);
            s1 = sigma1_32(X8);
            T1 = X10 += s0 + s1 + X3;
            ROUND_00_15(i + 10, g, h, a, b, c, d, e, f);

            s0 = sigma0_32(X12);
            s1 = sigma1_32(X9);
            T1 = X11 += s0 + s1 + X4;
            ROUND_00_15(i + 11, f, g, h, a, b, c, d, e);

            s0 = sigma0_32(X13);
            s1 = sigma1_32(X10);
            T1 = X12 += s0 + s1 + X5;
            ROUND_00_15(i + 12, e, f, g, h, a, b, c, d);

            s0 = sigma0_32(X14);
            s1 = sigma1_32(X11);
            T1 = X13 += s0 + s1 + X6;
            ROUND_00_15(i + 13, d, e, f, g, h, a, b, c);

            s0 = sigma0_32(X15);
            s1 = sigma1_32(X12);
            T1 = X14 += s0 + s1 + X7;
            ROUND_00_15(i + 14, c, d, e, f, g, h, a, b);

            s0 = sigma0_32(X0);
            s1 = sigma1_32(X13);
            T1 = X15 += s0 + s1 + X8;
            ROUND_00_15(i + 15, b, c, d, e, f, g, h, a);
        }

        state[0] += a;
        state[1] += b;
        state[2] += c;
        state[3] += d;
        state[4] += e;
        state[5] += f;
        state[6] += g;
        state[7] += h;
    }

    for (int i = 0; i < 8; i++)
        dev_store_bigendian_32(statebytes + 4 * i, state[i]);
}
#else // ifdef USING_SHA256_X_UNROLL

#define ROUND_16_63(i, a, b, c, d, e, f, g, h, X)                                                  \
    do {                                                                                           \
        s0 = X[(i + 1) & 0x0f];                                                                    \
        s0 = sigma0_32(s0);                                                                        \
        s1 = X[(i + 14) & 0x0f];                                                                   \
        s1 = sigma1_32(s1);                                                                        \
        T1 = X[(i) & 0x0f] += s0 + s1 + X[(i + 9) & 0x0f];                                         \
        ROUND_00_15(i, a, b, c, d, e, f, g, h);                                                    \
    } while (0)

__device__ void dev_crypto_hashblocks_sha256(uint8_t* statebytes, const void* in, size_t inlen) {
    u32 state[8];
    u32 a, b, c, d, e, f, g, h, s0, s1, T1;
    u32 X[16];
    u32 i;
    u32 num = inlen / 64;

    for (i = 0; i < 8; i++)
        state[i] = dev_load_bigendian_32(statebytes + 4 * i);

#ifdef USING_SHA256_INTEGER
    const u32* data = (const u32*) in;
#else  // ifdef USING_SHA256_INTEGER
    const u8* data = (const u8*) in;
#endif // ifdef USING_SHA256_INTEGER

    while (num--) {
        a = state[0];
        b = state[1];
        c = state[2];
        d = state[3];
        e = state[4];
        f = state[5];
        g = state[6];
        h = state[7];

        u32 l;

        (void) HOST_c2l(data, l);
        T1 = X[0] = l;
        ROUND_00_15(0, a, b, c, d, e, f, g, h);
        (void) HOST_c2l(data, l);
        T1 = X[1] = l;
        ROUND_00_15(1, h, a, b, c, d, e, f, g);
        (void) HOST_c2l(data, l);
        T1 = X[2] = l;
        ROUND_00_15(2, g, h, a, b, c, d, e, f);
        (void) HOST_c2l(data, l);
        T1 = X[3] = l;
        ROUND_00_15(3, f, g, h, a, b, c, d, e);
        (void) HOST_c2l(data, l);
        T1 = X[4] = l;
        ROUND_00_15(4, e, f, g, h, a, b, c, d);
        (void) HOST_c2l(data, l);
        T1 = X[5] = l;
        ROUND_00_15(5, d, e, f, g, h, a, b, c);
        (void) HOST_c2l(data, l);
        T1 = X[6] = l;
        ROUND_00_15(6, c, d, e, f, g, h, a, b);
        (void) HOST_c2l(data, l);
        T1 = X[7] = l;
        ROUND_00_15(7, b, c, d, e, f, g, h, a);
        (void) HOST_c2l(data, l);
        T1 = X[8] = l;
        ROUND_00_15(8, a, b, c, d, e, f, g, h);
        (void) HOST_c2l(data, l);
        T1 = X[9] = l;
        ROUND_00_15(9, h, a, b, c, d, e, f, g);
        (void) HOST_c2l(data, l);
        T1 = X[10] = l;
        ROUND_00_15(10, g, h, a, b, c, d, e, f);
        (void) HOST_c2l(data, l);
        T1 = X[11] = l;
        ROUND_00_15(11, f, g, h, a, b, c, d, e);
        (void) HOST_c2l(data, l);
        T1 = X[12] = l;
        ROUND_00_15(12, e, f, g, h, a, b, c, d);
        (void) HOST_c2l(data, l);
        T1 = X[13] = l;
        ROUND_00_15(13, d, e, f, g, h, a, b, c);
        (void) HOST_c2l(data, l);
        T1 = X[14] = l;
        ROUND_00_15(14, c, d, e, f, g, h, a, b);
        (void) HOST_c2l(data, l);
        T1 = X[15] = l;
        ROUND_00_15(15, b, c, d, e, f, g, h, a);

#pragma unroll 6
        for (i = 16; i < 64; i += 8) {
            ROUND_16_63(i + 0, a, b, c, d, e, f, g, h, X);
            ROUND_16_63(i + 1, h, a, b, c, d, e, f, g, X);
            ROUND_16_63(i + 2, g, h, a, b, c, d, e, f, X);
            ROUND_16_63(i + 3, f, g, h, a, b, c, d, e, X);
            ROUND_16_63(i + 4, e, f, g, h, a, b, c, d, X);
            ROUND_16_63(i + 5, d, e, f, g, h, a, b, c, X);
            ROUND_16_63(i + 6, c, d, e, f, g, h, a, b, X);
            ROUND_16_63(i + 7, b, c, d, e, f, g, h, a, X);
        }

        state[0] += a;
        state[1] += b;
        state[2] += c;
        state[3] += d;
        state[4] += e;
        state[5] += f;
        state[6] += g;
        state[7] += h;
    }

    for (i = 0; i < 8; i++)
        dev_store_bigendian_32(statebytes + 4 * i, state[i]);

} // sha256_block_data_order
#endif // ifdef USING_SHA256_X_UNROLL

#else // ifdef FASTER

__device__ void dev_crypto_hashblocks_sha256(uint8_t* statebytes, const void* in, size_t inlen) {
    u32 state[8];
    u32 a, b, c, d, e, f, g, h, s0, s1, T1, T2;
    u32 X[16], l;
    u32 i;
    u32 num = inlen / 64;

    for (i = 0; i < 8; i++)
        state[i] = dev_load_bigendian_32(statebytes + 4 * i);

#ifdef USING_SHA256_INTEGER
    const u32* data = (const u32*) in;
#else  // ifdef USING_SHA256_INTEGER
    const u8* data = (const u8*) in;
#endif // ifdef USING_SHA256_INTEGER

    while (num--) {
        a = state[0];
        b = state[1];
        c = state[2];
        d = state[3];
        e = state[4];
        f = state[5];
        g = state[6];
        h = state[7];

#ifdef USING_SHA256_UNROLL
#pragma unroll
#endif // ifdef USING_SHA256_UNROLL
        for (i = 0; i < 16; i++) {
            (void) HOST_c2l(data, l);
            T1 = X[i] = l;
            T1 += h + Sigma1_32(e) + Ch(e, f, g) + cons_K256[i];
            T2 = Sigma0_32(a) + Maj(a, b, c);
            h = g;
            g = f;
            f = e;
            e = d + T1;
            d = c;
            c = b;
            b = a;
            a = T1 + T2;
        }

#ifdef USING_SHA256_UNROLL
#pragma unroll
#endif // ifdef USING_SHA256_UNROLL
        for (i = 16; i < 64; i++) {
            s0 = X[(i + 1) & 0x0f];
            s0 = sigma0_32(s0);
            s1 = X[(i + 14) & 0x0f];
            s1 = sigma1_32(s1);

            T1 = X[i & 0xf] += s0 + s1 + X[(i + 9) & 0xf];
            T1 += h + Sigma1_32(e) + Ch(e, f, g) + cons_K256[i];
            T2 = Sigma0_32(a) + Maj(a, b, c);
            h = g;
            g = f;
            f = e;
            e = d + T1;
            d = c;
            c = b;
            b = a;
            a = T1 + T2;
        }

        state[0] += a;
        state[1] += b;
        state[2] += c;
        state[3] += d;
        state[4] += e;
        state[5] += f;
        state[6] += g;
        state[7] += h;
    }

    for (i = 0; i < 8; i++)
        dev_store_bigendian_32(statebytes + 4 * i, state[i]);
} // dev_crypto_hashblocks_sha256

#endif // ifdef FASTER

__device__ void dev_sha256_inc_init(uint8_t* state) {
    u8 iv[40] = {0x6a, 0x09, 0xe6, 0x67, 0xbb, 0x67, 0xae, 0x85, 0x3c, 0x6e, 0xf3, 0x72, 0xa5, 0x4f,
                 0xf5, 0x3a, 0x51, 0x0e, 0x52, 0x7f, 0x9b, 0x05, 0x68, 0x8c, 0x1f, 0x83, 0xd9, 0xab,
                 0x5b, 0xe0, 0xcd, 0x19, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00};

    memcpy(state, iv, 40);
} // dev_sha256_inc_init

__device__ void dev_sha256_inc_blocks(uint8_t* state, const void* in, size_t inblocks) {
    if (inblocks == 0) return;

    uint64_t bytes = dev_load_bigendian_64(state + 32);

    dev_crypto_hashblocks_sha256(state, in, 64 * inblocks);
    bytes += 64 * inblocks;

    dev_store_bigendian_64(state + 32, bytes);
} // dev_sha256_inc_blocks

__device__ void dev_sha256_inc_finalize(uint8_t* out, uint8_t* state, void* in_, size_t inlen) {
    // u8 padded[128];
    uint32_t padded_[32];
    u8* padded = (u8*) padded_;

    memset(padded, 0, 128);
    uint64_t bytes = dev_load_bigendian_64(state + 32) + inlen;

    u8* in = (u8*) in_;

    dev_crypto_hashblocks_sha256(state, in, inlen);

    in += inlen;
    inlen &= 63;
    in -= inlen;

    if (inlen != 0) memcpy(padded, in, inlen);

    padded[inlen] = 0x80;

    u32 bytes_arr[8] = {53, 45, 37, 29, 21, 13, 5, 3};

    if (inlen < 56) {
        memset(padded + inlen + 1, 0, 56 - inlen - 1);
        // padded[63] = (uint8_t)(bytes << 3);
        // padded[56] = (uint8_t)(bytes >> 53);
        // padded[57] = (uint8_t)(bytes >> 45);
        // padded[58] = (uint8_t)(bytes >> 37);
        // padded[59] = (uint8_t)(bytes >> 29);
        // padded[60] = (uint8_t)(bytes >> 21);
        // padded[61] = (uint8_t)(bytes >> 13);
        // padded[62] = (uint8_t)(bytes >> 5);
        // padded[63] = (uint8_t)(bytes << 3);
        for (size_t i = 0; i < 7; i++)
            padded[i + 56] = (uint8_t) (bytes >> bytes_arr[i]);
        padded[63] = (uint8_t) (bytes << 3);

        dev_crypto_hashblocks_sha256(state, (void*) padded, 64);
    } else {
        memset(in + inlen + 1, 0, 120 - inlen - 1);
        padded[120] = (uint8_t) (bytes >> 53);
        padded[121] = (uint8_t) (bytes >> 45);
        padded[122] = (uint8_t) (bytes >> 37);
        padded[123] = (uint8_t) (bytes >> 29);
        padded[124] = (uint8_t) (bytes >> 21);
        padded[125] = (uint8_t) (bytes >> 13);
        padded[126] = (uint8_t) (bytes >> 5);
        padded[127] = (uint8_t) (bytes << 3);
        // for (size_t i = 0; i < 8; i++)
        // 	padded[i + 120] = (uint8_t)(bytes >> bytes_arr[i]);
        dev_crypto_hashblocks_sha256(state, (void*) padded, 128);
    }
    memcpy(out, state, 32);
}

__device__ void dev_sha256(uint8_t* out, uint8_t* in, size_t inlen) {
    uint8_t state[40];
    static u8 m[32];

    // if (out == NULL) out = m;

    dev_sha256_inc_init(state);
    dev_sha256_inc_finalize(out, state, in, inlen);
}

__global__ void global_sha256(uint8_t* out, uint8_t* in, size_t inlen, size_t loop_num) {
    for (int i = 0; i < loop_num; i++)
        dev_sha256(out, in, inlen);
} // global_sha256

void face_sha256(uint8_t* md, uint8_t* d, size_t n, size_t loop_num) {
    struct timespec start, stop;
    CHECK(hipSetDevice(DEVICE_USED));
    u8 *dev_d = NULL, *dev_md = NULL;

    CHECK(hipMalloc((void**) &dev_d, n * sizeof(u8)));
    CHECK(hipMalloc((void**) &dev_md, 32 * sizeof(u8)));
    CHECK(hipMemcpy(dev_d, d, n * sizeof(u8), HOST_2_DEVICE));

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);

    CHECK(hipDeviceSynchronize());
    global_sha256<<<1, 1>>>(dev_md, dev_d, n, loop_num);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);

    g_result = (stop.tv_sec - start.tv_sec) * 1e6 + (stop.tv_nsec - start.tv_nsec) / 1e3;

    CHECK(hipMemcpy(md, dev_md, 32 * sizeof(u8), DEVICE_2_HOST));

    hipFree(dev_d);
    hipFree(dev_md);
}

__global__ void global_dp_sha256(uint8_t* out, const uint8_t* in, size_t inlen, size_t total_msg_num) {
    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= total_msg_num) return;

    // Calculate offset for this thread's input and output
    uint8_t* my_out = out + tid * 32;  // Each hash output is 32 bytes
    const uint8_t* my_in = in + tid * inlen;
    
    dev_sha256(my_out, (uint8_t*)my_in, inlen);
}

void face_dp_sha256(const uint8_t* in, uint8_t* out, size_t msg_size,
                    size_t total_msg_num, size_t grid_size, size_t block_size) {
    struct timespec start, stop;
    CHECK(hipSetDevice(DEVICE_USED));
    
    uint8_t *dev_in = NULL, *dev_out = NULL;
    size_t total_in_size = msg_size * total_msg_num;
    size_t total_out_size = 32 * total_msg_num;  // 32 bytes per SHA256 hash

    // Allocate device memory
    CHECK(hipMalloc((void**)&dev_in, total_in_size));
    CHECK(hipMalloc((void**)&dev_out, total_out_size));

    // Copy input data to device
    CHECK(hipMemcpy(dev_in, in, total_in_size, HOST_2_DEVICE));

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);

    // Launch kernel with specified grid and block sizes
    CHECK(hipDeviceSynchronize());
    global_dp_sha256<<<grid_size, block_size>>>(dev_out, dev_in, msg_size, total_msg_num);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());

    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
    g_result = (stop.tv_sec - start.tv_sec) * 1e6 + (stop.tv_nsec - start.tv_nsec) / 1e3;

    // Copy results back to host
    CHECK(hipMemcpy(out, dev_out, total_out_size, DEVICE_2_HOST));

    // Clean up
    hipFree(dev_in);
    hipFree(dev_out);
}

/**
 * Note that inlen should be sufficiently small that it still allows for
 * an array to be allocated on the stack. Typically 'in' is merely a seed.
 * Outputs outlen number of bytes
 */
__device__ void dev_mgf1(unsigned char* out, unsigned long outlen, const unsigned char* in,
                         unsigned long inlen) {
    unsigned char inbuf[SPX_N + SPX_SHA256_ADDR_BYTES + 4];
    // unsigned char outbuf[SPX_SHA256_OUTPUT_BYTES]; // 715 wrong
    unsigned char outbuf[SPX_SHA256_OUTPUT_BYTES * 2];
    u32 i;

    memcpy(inbuf, in, inlen);

    /* While we can fit in at least another full block of SHA256 output.. */
    for (i = 0; (i + 1) * SPX_SHA256_OUTPUT_BYTES <= outlen; i++) {
        dev_u32_to_bytes(inbuf + inlen, i);
        dev_sha256(out, inbuf, inlen + 4);
        out += SPX_SHA256_OUTPUT_BYTES;
    }
    /* Until we cannot anymore, and we fill the remainder. */
    if (outlen > i * SPX_SHA256_OUTPUT_BYTES) {
        dev_u32_to_bytes(inbuf + inlen, i);
        dev_sha256(outbuf, inbuf, inlen + 4);
        memcpy(out, outbuf, outlen - i * SPX_SHA256_OUTPUT_BYTES);
    }

    // const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    //
    // out -= SPX_SHA256_OUTPUT_BYTES;
    // if (tid == 0) {
    // 	printf("gpu\n");
    // 	printf("out = %02x\n", out[0]);
    // 	printf("out = %02x\n", out[1]);
    // 	printf("out = %02x\n", out[2]);
    // }
    // int a = 1;
    //
    // while (a) {
    // }

} // dev_mgf1

__device__ void dev_mgf1_hg(unsigned char* out, unsigned long outlen, const unsigned char* in,
                            unsigned long inlen) {
    unsigned char inbuf[SPX_SHA256_OUTPUT_BYTES + 4]; // inlen + 4
    unsigned char outbuf[SPX_SHA256_OUTPUT_BYTES];
    unsigned long i;

    memcpy(inbuf, in, inlen);

    /* While we can fit in at least another full block of SHA256 output.. */
    for (i = 0; (i + 1) * SPX_SHA256_OUTPUT_BYTES <= outlen; i++) {
        dev_u32_to_bytes(inbuf + inlen, i);
        dev_sha256(out, inbuf, inlen + 4);
        out += SPX_SHA256_OUTPUT_BYTES;
    }
    /* Until we cannot anymore, and we fill the remainder. */
    if (outlen > i * SPX_SHA256_OUTPUT_BYTES) {
        dev_u32_to_bytes(inbuf + inlen, i);
        dev_sha256(outbuf, inbuf, inlen + 4);
        memcpy(out, outbuf, outlen - i * SPX_SHA256_OUTPUT_BYTES);
    }
    // if (outlen / SPX_SHA256_OUTPUT_BYTES > 1) no output
    // 	printf("outlen = %d\n", outlen / SPX_SHA256_OUTPUT_BYTES);

} // dev_mgf1_hg

__device__ uint8_t dev_state_seeded[40];

/**
 * Absorb the constant pub_seed using one round of the compression function
 * This initializes state_seeded, which can then be reused in thash
 **/
__device__ void dev_seed_state(const unsigned char* pub_seed) {
    uint8_t block[SPX_SHA256_BLOCK_BYTES];
    size_t i;

    for (i = 0; i < SPX_N; ++i) {
        block[i] = pub_seed[i];
    }
    for (i = SPX_N; i < SPX_SHA256_BLOCK_BYTES; ++i) {
        block[i] = 0;
    }

    dev_sha256_inc_init(dev_state_seeded);
    dev_sha256_inc_blocks(dev_state_seeded, block, 1);
} // seed_state

#endif // ifdef SHA256