// #define _POSIX_C_SOURCE 199309L

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "api.h"
#include "common.h"
#include "fors.h"
#include "params.h"
#include "rng.h"
#include "wots.h"

#define SPX_MLEN 32
#define NTESTS 5

// multi-keypair is tested only in paper

#include "all_option.h"

#define MEASURE_WHOLE(TEXT, MUL, FNCALL)                                                           \
    printf(TEXT);                                                                                  \
    g_result = 0;                                                                                  \
    g_count = 0;                                                                                   \
    for (int i = 0; i < MUL; i++)                                                                  \
        FNCALL;                                                                                    \
    printf("whole: %11.2lf ms (%2.2lf sec)\n", g_result / MUL / 1e3, g_result / MUL / 1e6);

#define MEASURE_INNER(TEXT, MUL, FNCALL)                                                           \
    printf(TEXT);                                                                                  \
    g_inner_result = 0;                                                                            \
    g_count = 0;                                                                                   \
    for (int i = 0; i < MUL; i++)                                                                  \
        FNCALL;                                                                                    \
    printf("inner: %11.2lf ms (%2.2lf sec)\n", g_inner_result / MUL / 1e3,                         \
           g_inner_result / MUL / 1e6);

int main(int argc, char** argv) {
    /* Make stdout buffer more responsive. */
    setbuf(stdout, NULL);

    int num = 65536;

    if (argv[1] != NULL) num = atoi(argv[1]);

    u8 *pk, *sk;
    u8 *m, *sm, *mout;

    CHECK(hipHostMalloc(&pk, SPX_PK_BYTES * num));
    CHECK(hipHostMalloc(&sk, SPX_SK_BYTES * num));
    CHECK(hipHostMalloc(&m, SPX_MLEN * num));
    CHECK(hipHostMalloc(&sm, (SPX_BYTES + SPX_MLEN) * num));
    CHECK(hipHostMalloc(&mout, SPX_MLEN * num));

    unsigned long long smlen;
    unsigned long long mlen;

    randombytes(m, SPX_MLEN * num);

    show_para();
    printf("num = %d\n", num);
    printf("Parameters: n = %d, h = %d, d = %d, a = %d, k = %d, w = %d, len = %d\n", SPX_N,
           SPX_FULL_HEIGHT, SPX_D, SPX_FORS_HEIGHT, SPX_FORS_TREES, SPX_WOTS_W, SPX_WOTS_LEN);

    printf("warming up 1 iter\n");
    for (int i = 0; i < 1; i++) {
        face_mdp_crypto_sign_keypair(pk, sk, num, 512, 32);
        face_mdp_crypto_sign(sm, &smlen, m, SPX_MLEN, sk, num, 512, 32);
        face_mdp_crypto_sign_open(mout, &mlen, sm, smlen, pk, num, 512, 32);
    }
    printf("Running %d iterations.\n", NTESTS);

    // Test with fixed number of keypairs (32768) and different block/thread configurations
    int test_num = 32768;

    // Test different block and thread configurations
    // printf("multi-keypair data parallelism with different configurations on %d number task\n",
    //        test_num);
    // printf("blocks, threads, time(ms), per op(ms)\n");

    // Test different thread counts
    int block_counts[] = {32, 64, 96, 128, 160, 192, 224, 256};
    int thread_counts[] = {64, 96, 128, 160, 192, 224, 256, 320, 384, 512};

    // for (int thread_idx = 0; thread_idx < sizeof(thread_counts) / sizeof(int); thread_idx++) {
    //     int threads = thread_counts[thread_idx];

    //     for (int block_idx = 0; block_idx < sizeof(block_counts) / sizeof(int); block_idx++) {
    //         int blocks = block_counts[block_idx];

    //         g_result = 0;
    //         for (int j = 0; j < NTESTS; j++) {
    //             face_mdp_crypto_sign_keypair(pk, sk, test_num, blocks, threads);
    //         }
    //         double avg_time = g_result / NTESTS / 1e3; // convert to ms
    //         printf("%d, %d, %.2lf, %.4lf\n", blocks, threads, avg_time, avg_time / test_num);
    //     }
    // }

    // Test sign with fixed number of signatures (32768) and different block/thread configurations
    // printf("\nmulti-signature data parallelism with different configurations on %d number
    // task\n",
    //        test_num);
    // printf("blocks, threads, time(ms), per op(ms)\n");

    // for (int thread_idx = 0; thread_idx < sizeof(thread_counts) / sizeof(int); thread_idx++) {
    //     int threads = thread_counts[thread_idx];

    //     for (int block_idx = 0; block_idx < sizeof(block_counts) / sizeof(int); block_idx++) {
    //         int blocks = block_counts[block_idx];

    //         g_result = 0;
    //         for (int j = 0; j < NTESTS; j++) {
    //             face_mdp_crypto_sign(sm, &smlen, m, SPX_MLEN, sk, test_num, blocks, threads);
    //         }
    //         double avg_time = g_result / NTESTS / 1e3; // convert to ms
    //         printf("%d, %d, %.2lf, %.4lf\n", blocks, threads, avg_time, avg_time / test_num);
    //     }
    // }

    // Test sign_open with fixed number of signatures (32768) and different block/thread
    // configurations
    printf(
        "\nmulti-verification data parallelism with different configurations on %d number task\n",
        test_num);
    printf("blocks, threads, time(ms), per op(ms)\n");

    for (int thread_idx = 0; thread_idx < sizeof(thread_counts) / sizeof(int); thread_idx++) {
        int threads = thread_counts[thread_idx];

        for (int block_idx = 0; block_idx < sizeof(block_counts) / sizeof(int); block_idx++) {
            int blocks = block_counts[block_idx];

            g_result = 0;
            for (int j = 0; j < NTESTS; j++) {
                face_mdp_crypto_sign_open(mout, &mlen, sm, smlen, pk, test_num, blocks, threads);
            }
            double avg_time = g_result / NTESTS / 1e3; // convert to ms
            printf("%d, %d, %.2lf, %.4lf\n", blocks, threads, avg_time, avg_time / test_num);
        }
    }

    // Original scaling test
    // printf("\nmulti-keypair data parallelism scaling test on 512*32 block*thread\n");
    // printf("number, keypair(ms), keypair per op(ms)\n");
    // for (int i = 1024; i <= 65536; i *= 2) {
    //     double t1;
    //     g_result = 0;
    //     for (int j = 0; j < NTESTS; j++)
    //         face_mdp_crypto_sign_keypair(pk, sk, i, 512, 32);
    //     t1 = g_result / NTESTS / 1e3;
    //     printf("%d, %.2lf, %.4lf\n", i, t1, t1 / i);
    // }

    CHECK(hipHostFree(pk));
    CHECK(hipHostFree(sk));
    CHECK(hipHostFree(m));
    CHECK(hipHostFree(sm));
    CHECK(hipHostFree(mout));

    return 0;
} // main
