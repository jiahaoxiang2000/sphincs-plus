// #define _POSIX_C_SOURCE 199309L

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "api.h"
#include "common.h"
#include "fors.h"
#include "params.h"
#include "rng.h"
#include "wots.h"

#define SPX_MLEN 32
#define NTESTS 5

// multi-keypair is tested only in paper

#include "all_option.h"

#define MEASURE_WHOLE(TEXT, MUL, FNCALL)                                                           \
    printf(TEXT);                                                                                  \
    g_result = 0;                                                                                  \
    g_count = 0;                                                                                   \
    for (int i = 0; i < MUL; i++)                                                                  \
        FNCALL;                                                                                    \
    printf("whole: %11.2lf ms (%2.2lf sec)\n", g_result / MUL / 1e3, g_result / MUL / 1e6);

#define MEASURE_INNER(TEXT, MUL, FNCALL)                                                           \
    printf(TEXT);                                                                                  \
    g_inner_result = 0;                                                                            \
    g_count = 0;                                                                                   \
    for (int i = 0; i < MUL; i++)                                                                  \
        FNCALL;                                                                                    \
    printf("inner: %11.2lf ms (%2.2lf sec)\n", g_inner_result / MUL / 1e3,                         \
           g_inner_result / MUL / 1e6);

int main(int argc, char** argv) {
    /* Make stdout buffer more responsive. */
    setbuf(stdout, NULL);

    int num = 65536;

    if (argv[1] != NULL) num = atoi(argv[1]);

    u8 *pk, *sk;
    u8 *m, *sm, *mout;

    CHECK(hipHostMalloc(&pk, SPX_PK_BYTES * num));
    CHECK(hipHostMalloc(&sk, SPX_SK_BYTES * num));
    CHECK(hipHostMalloc(&m, SPX_MLEN * num));
    CHECK(hipHostMalloc(&sm, (SPX_BYTES + SPX_MLEN) * num));
    CHECK(hipHostMalloc(&mout, SPX_MLEN * num));

    unsigned long long smlen;
    unsigned long long mlen;

    randombytes(m, SPX_MLEN * num);

    show_para();
    printf("num = %d\n", num);
    printf("Parameters: n = %d, h = %d, d = %d, a = %d, k = %d, w = %d, len = %d\n", SPX_N,
           SPX_FULL_HEIGHT, SPX_D, SPX_FORS_HEIGHT, SPX_FORS_TREES, SPX_WOTS_W, SPX_WOTS_LEN);

    printf("warming up 1 iter\n");
    for (int i = 0; i < 1; i++) {
        face_mdp_crypto_sign_keypair(pk, sk, num, 512, 32);
        // face_mdp_crypto_sign(sm, &smlen, m, SPX_MLEN, sk, num);
        // face_mdp_crypto_sign_open(mout, &mlen, sm, smlen, pk, num);
    }
    printf("Running %d iterations.\n", NTESTS);

    printf("multi-keypair data parallelism\n");
    printf("number, keypair, sign, verify, keypair per op, Sign per op, verify per op\n");
    for (int i = 1024; i <= 65536; i = i * 2) {
        double t1, t2, t3;
        g_result = 0;
        for (int j = 0; j < NTESTS; j++)
            face_mdp_crypto_sign_keypair(pk, sk, i, 512, 32);
        t1 = g_result / NTESTS / 1e3;
        g_result = 0;
        printf("%d, %.2lf, %.2lf, %.2lf, %.4lf, %.4lf, %.4lf\n", i, t1, t2, t3, t1 / i, t2 / i,
               t3 / i);
    }

    CHECK(hipHostFree(pk));
    CHECK(hipHostFree(sk));
    CHECK(hipHostFree(m));
    CHECK(hipHostFree(sm));
    CHECK(hipHostFree(mout));

    return 0;
} // main
