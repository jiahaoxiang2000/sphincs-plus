#include "hip/hip_runtime.h"
// #define _POSIX_C_SOURCE 199309L

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "api.h"
#include "common.h"
#include "fors.h"
#include "params.h"
#include "rng.h"
#include "wots.h"

#define SPX_MLEN 32
#define NTESTS 5

// multi-keypair is tested only in paper

#include "all_option.h"

// Function to run performance test and return metrics
double run_perf_test(int num_tasks, int blocks, int threads, int operation, u8* pk, u8* sk, u8* m,
                     u8* sm, u8* mout, unsigned long long* smlen, unsigned long long* mlen) {
    g_result = 0;
    for (int j = 0; j < NTESTS; j++) {
        switch (operation) {
        case 0: // Keypair
            face_mdp_crypto_sign_keypair(pk, sk, num_tasks, blocks, threads);
            break;
        case 1: // Sign
            face_mdp_crypto_sign(sm, smlen, m, SPX_MLEN, sk, num_tasks, blocks, threads);
            break;
        case 2: // Verify
            face_mdp_crypto_sign_open(mout, mlen, sm, *smlen, pk, num_tasks, blocks, threads);
            break;
        }
    }
    return g_result / NTESTS / 1e3; // convert to ms
}

int main(int argc, char** argv) {
    /* Make stdout buffer more responsive. */
    setbuf(stdout, NULL);

    int num = 32768;

    if (argv[1] != NULL) num = atoi(argv[1]);

    u8 *pk, *sk;
    u8 *m, *sm, *mout;

    CHECK(hipHostMalloc(&pk, SPX_PK_BYTES * num));
    CHECK(hipHostMalloc(&sk, SPX_SK_BYTES * num));
    CHECK(hipHostMalloc(&m, SPX_MLEN * num));
    CHECK(hipHostMalloc(&sm, (SPX_BYTES + SPX_MLEN) * num));
    CHECK(hipHostMalloc(&mout, SPX_MLEN * num));

    unsigned long long smlen;
    unsigned long long mlen;

    randombytes(m, SPX_MLEN * num);

    show_para();
    printf("num = %d\n", num);
    printf("Parameters: n = %d, h = %d, d = %d, a = %d, k = %d, w = %d, len = %d\n", SPX_N,
           SPX_FULL_HEIGHT, SPX_D, SPX_FORS_HEIGHT, SPX_FORS_TREES, SPX_WOTS_W, SPX_WOTS_LEN);

    printf("warming up 1 iter\n");
    for (int i = 0; i < 1; i++) {
        face_mdp_crypto_sign_keypair(pk, sk, num, 512, 32);
        face_mdp_crypto_sign(sm, &smlen, m, SPX_MLEN, sk, num, 512, 32);
        face_mdp_crypto_sign_open(mout, &mlen, sm, smlen, pk, num, 512, 32);
    }
    printf("Running %d iterations.\n", NTESTS);

    // Test with fixed number of keypairs (32768) and different block/thread configurations
    int test_num = num;

    // Define original and optimized configurations
    int original_blocks = 512;
    int original_threads = 32;

    // Optimal configurations from parameter.csv
    int optimal_blocks_kg = 128;
    int optimal_threads_kg = 256;
    int optimal_blocks_sign = 160;
    int optimal_threads_sign = 256;
    int optimal_blocks_verify = 128;
    int optimal_threads_verify = 256;

    // Print CSV header
    printf("\nPerformance Comparison (CSV format)\n");
    printf(
        "Configuration,KG Latency (ms),Sign Latency (ms),Verify Latency (ms),KG Throughput "
        "(tasks/sec),Sign Throughput (tasks/sec),Verify Throughput (tasks/sec)\n");

    // Test original configuration (512 blocks, 32 threads)
    double kg_latency_orig = run_perf_test(test_num, original_blocks, original_threads, 0, pk, sk,
                                           m, sm, mout, &smlen, &mlen);
    double sign_latency_orig = run_perf_test(test_num, original_blocks, original_threads, 1, pk, sk,
                                             m, sm, mout, &smlen, &mlen);
    double verify_latency_orig = run_perf_test(test_num, original_blocks, original_threads, 2, pk,
                                               sk, m, sm, mout, &smlen, &mlen);

    // much round run
    int original_round_number = test_num / (original_blocks * original_threads) + 1;
    kg_latency_orig = kg_latency_orig / original_round_number;
    sign_latency_orig = sign_latency_orig / original_round_number;
    verify_latency_orig = verify_latency_orig / original_round_number;
    double kg_throughput_orig = test_num / (kg_latency_orig / 1000.0);
    double sign_throughput_orig = test_num / (sign_latency_orig / 1000.0);
    double verify_throughput_orig = test_num / (verify_latency_orig / 1000.0);

    printf("Original (512x32),%.2f,%.2f,%.2f,%.2f,%.2f,%.2f\n", kg_latency_orig, sign_latency_orig,
           verify_latency_orig, kg_throughput_orig, sign_throughput_orig, verify_throughput_orig);

    // Test optimized configuration from parameter.csv
    double kg_latency_opt = run_perf_test(test_num, optimal_blocks_kg, optimal_threads_kg, 0, pk,
                                          sk, m, sm, mout, &smlen, &mlen);
    double sign_latency_opt = run_perf_test(test_num, optimal_blocks_sign, optimal_threads_sign, 1,
                                            pk, sk, m, sm, mout, &smlen, &mlen);
    double verify_latency_opt
        = run_perf_test(test_num, optimal_blocks_verify, optimal_threads_verify, 2, pk, sk, m, sm,
                        mout, &smlen, &mlen);

    // Calculate round numbers for optimized configurations
    int kg_round_number = test_num / (optimal_blocks_kg * optimal_threads_kg) + 1;
    int sign_round_number = test_num / (optimal_blocks_sign * optimal_threads_sign) + 1;
    int verify_round_number = test_num / (optimal_blocks_verify * optimal_threads_verify) + 1;

    // Adjust latency by round number
    kg_latency_opt = kg_latency_opt / original_round_number;
    sign_latency_opt = sign_latency_opt / original_round_number;
    verify_latency_opt = verify_latency_opt / original_round_number;

    double kg_throughput_opt = test_num / (kg_latency_opt / 1000.0);
    double sign_throughput_opt = test_num / (sign_latency_opt / 1000.0);
    double verify_throughput_opt = test_num / (verify_latency_opt / 1000.0);

    printf("Optimized,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f\n", kg_latency_opt, sign_latency_opt,
           verify_latency_opt, kg_throughput_opt, sign_throughput_opt, verify_throughput_opt);

    // Calculate improvement percentage
    double kg_improvement = (kg_latency_orig - kg_latency_opt) / kg_latency_orig * 100.0;
    double sign_improvement = (sign_latency_orig - sign_latency_opt) / sign_latency_orig * 100.0;
    double verify_improvement
        = (verify_latency_orig - verify_latency_opt) / verify_latency_orig * 100.0;

    printf("\nImprovement Percentage (%%)\n");
    printf("KG: %.2f%%, Sign: %.2f%%, Verify: %.2f%%\n", kg_improvement, sign_improvement,
           verify_improvement);

    CHECK(hipHostFree(pk));
    CHECK(hipHostFree(sk));
    CHECK(hipHostFree(m));
    CHECK(hipHostFree(sm));
    CHECK(hipHostFree(mout));

    return 0;
} // main
