// #define _POSIX_C_SOURCE 199309L

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "api.h"
#include "common.h"
#include "fors.h"
#include "params.h"
#include "rng.h"
#include "wots.h"

#define SPX_MLEN 32
#define NTESTS 5

// multi-keypair is tested only in paper

#include "all_option.h"


int main(int argc, char** argv) {
    /* Make stdout buffer more responsive. */
    setbuf(stdout, NULL);

    int num = 32768;

    if (argv[1] != NULL) num = atoi(argv[1]);

    u8 *pk, *sk;
    u8 *m, *sm, *mout;

    CHECK(hipHostMalloc(&pk, SPX_PK_BYTES * num));
    CHECK(hipHostMalloc(&sk, SPX_SK_BYTES * num));
    CHECK(hipHostMalloc(&m, SPX_MLEN * num));
    CHECK(hipHostMalloc(&sm, (SPX_BYTES + SPX_MLEN) * num));
    CHECK(hipHostMalloc(&mout, SPX_MLEN * num));

    unsigned long long smlen;
    unsigned long long mlen;

    randombytes(m, SPX_MLEN * num);

    show_para();
    printf("num = %d\n", num);
    printf("Parameters: n = %d, h = %d, d = %d, a = %d, k = %d, w = %d, len = %d\n", SPX_N,
           SPX_FULL_HEIGHT, SPX_D, SPX_FORS_HEIGHT, SPX_FORS_TREES, SPX_WOTS_W, SPX_WOTS_LEN);

    printf("warming up 1 iter\n");
    for (int i = 0; i < 1; i++) {
        face_mdp_crypto_sign_keypair(pk, sk, num, 512, 32);
        face_mdp_crypto_sign(sm, &smlen, m, SPX_MLEN, sk, num, 512, 32);
        face_mdp_crypto_sign_open(mout, &mlen, sm, smlen, pk, num, 512, 32);
    }
    printf("Running %d iterations.\n", NTESTS);

    // Test with fixed number of keypairs (32768) and different block/thread configurations
    int test_num = num;

    // Determine which SPHINCS+ variant is being used
    const char* variant;
    #ifdef SPX_128S
        variant = "128S";
    #elif defined(SPX_128F)
        variant = "128F";
    #elif defined(SPX_192S)
        variant = "192S";
    #elif defined(SPX_192F)
        variant = "192F";
    #elif defined(SPX_256S)
        variant = "256S";
    #elif defined(SPX_256F)
        variant = "256F";
    #endif

    // Test different block and thread configurations
    int block_counts[] = {32, 64, 96, 128, 160, 192, 224, 256};
    int thread_counts[] = {64, 96, 128, 160, 192, 224, 256, 320, 384, 512};

    printf("\n%s-SLH-DSA-%d\n", variant, test_num);
    printf("function,blocks,threads,time(ms),per_op(ms)\n");
    
    for (int thread_idx = 0; thread_idx < sizeof(thread_counts) / sizeof(int); thread_idx++) {
        int threads = thread_counts[thread_idx];

        for (int block_idx = 0; block_idx < sizeof(block_counts) / sizeof(int); block_idx++) {
            int blocks = block_counts[block_idx];

            g_result = 0;
            for (int j = 0; j < NTESTS; j++) {
                face_mdp_crypto_sign_keypair(pk, sk, test_num, blocks, threads);
            }
            double avg_time = g_result / NTESTS / 1e3; // convert to ms
            printf("%s-keypair,%d,%d,%.2lf,%.4lf\n", variant, blocks, threads, avg_time, avg_time / test_num);
        }
    }
    

    for (int thread_idx = 0; thread_idx < sizeof(thread_counts) / sizeof(int); thread_idx++) {
        int threads = thread_counts[thread_idx];

        for (int block_idx = 0; block_idx < sizeof(block_counts) / sizeof(int); block_idx++) {
            int blocks = block_counts[block_idx];

            g_result = 0;
            for (int j = 0; j < NTESTS; j++) {
                face_mdp_crypto_sign(sm, &smlen, m, SPX_MLEN, sk, test_num, blocks, threads);
            }
            double avg_time = g_result / NTESTS / 1e3; // convert to ms
            printf("%s-sign,%d,%d,%.2lf,%.4lf\n", variant, blocks, threads, avg_time, avg_time / test_num);
        }
    }
    

    for (int thread_idx = 0; thread_idx < sizeof(thread_counts) / sizeof(int); thread_idx++) {
        int threads = thread_counts[thread_idx];

        for (int block_idx = 0; block_idx < sizeof(block_counts) / sizeof(int); block_idx++) {
            int blocks = block_counts[block_idx];

            g_result = 0;
            for (int j = 0; j < NTESTS; j++) {
                face_mdp_crypto_sign_open(mout, &mlen, sm, smlen, pk, test_num, blocks, threads);
            }
            double avg_time = g_result / NTESTS / 1e3; // convert to ms
            printf("%s-verify,%d,%d,%.2lf,%.4lf\n", variant, blocks, threads, avg_time, avg_time / test_num);
        }
    }

    CHECK(hipHostFree(pk));
    CHECK(hipHostFree(sk));
    CHECK(hipHostFree(m));
    CHECK(hipHostFree(sm));
    CHECK(hipHostFree(mout));

    return 0;
} // main